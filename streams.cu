#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

#define N 100000000
#define THREADS_PER_BLOCK 256

void checkCudaError(hipError_t err, const char *msg) {
  if (err != hipSuccess) {
    std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err)
              << std::endl;
    exit(EXIT_FAILURE);
  }
}

__global__ void simpleKernel1(float *data) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    data[idx] += 1.0f;
  }
}

__global__ void simpleKernel2(float *data) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    data[idx] *= 2.0f;
  }
}

void singleStreamTransferAndCompute(float *h_data, float *d_data1,
                                    float *d_data2) {
  hipStream_t stream;
  checkCudaError(hipStreamCreate(&stream), "Creating stream");

  // Transfer data to GPU1
  checkCudaError(hipSetDevice(0), "Setting device 0");
  checkCudaError(hipMemcpyAsync(d_data1, h_data, N * sizeof(float),
                                 hipMemcpyHostToDevice, stream),
                 "MemcpyAsync to GPU1");

  // Transfer data to GPU2
  checkCudaError(hipSetDevice(1), "Setting device 1");
  checkCudaError(hipMemcpyAsync(d_data2, h_data, N * sizeof(float),
                                 hipMemcpyHostToDevice, stream),
                 "MemcpyAsync to GPU2");

  // Synchronize the stream
  checkCudaError(hipStreamSynchronize(stream), "Stream synchronize");

  // Launch kernels on GPU1
  checkCudaError(hipSetDevice(0), "Setting device 0");
  simpleKernel1<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK,
                  THREADS_PER_BLOCK>>>(d_data1);
  checkCudaError(hipStreamSynchronize(stream),
                 "Stream synchronize after kernel1 on GPU1");
  simpleKernel2<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK,
                  THREADS_PER_BLOCK>>>(d_data1);
  checkCudaError(hipStreamSynchronize(stream),
                 "Stream synchronize after kernel2 on GPU1");

  // Launch kernels on GPU2
  checkCudaError(hipSetDevice(1), "Setting device 1");
  simpleKernel1<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK,
                  THREADS_PER_BLOCK>>>(d_data2);
  checkCudaError(hipStreamSynchronize(stream),
                 "Stream synchronize after kernel1 on GPU2");
  simpleKernel2<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK,
                  THREADS_PER_BLOCK>>>(d_data2);
  checkCudaError(hipStreamSynchronize(stream),
                 "Stream synchronize after kernel2 on GPU2");

  // Clean up
  checkCudaError(hipStreamDestroy(stream), "Destroying stream");
}

void multiStreamTransferAndCompute(float *h_data, float *d_data1,
                                   float *d_data2) {
  hipStream_t stream1, stream2;
  checkCudaError(hipStreamCreate(&stream1), "Creating stream1");
  checkCudaError(hipStreamCreate(&stream2), "Creating stream2");

  // Transfer data to GPU1 using stream1
  checkCudaError(hipSetDevice(0), "Setting device 0");
  checkCudaError(hipMemcpyAsync(d_data1, h_data, N * sizeof(float),
                                 hipMemcpyHostToDevice, stream1),
                 "MemcpyAsync to GPU1");

  // Transfer data to GPU2 using stream2
  checkCudaError(hipSetDevice(1), "Setting device 1");
  checkCudaError(hipMemcpyAsync(d_data2, h_data, N * sizeof(float),
                                 hipMemcpyHostToDevice, stream2),
                 "MemcpyAsync to GPU2");

  // Launch kernels on GPU1 using stream1
  checkCudaError(hipSetDevice(0), "Setting device 0");
  simpleKernel1<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK,
                  THREADS_PER_BLOCK, 0, stream1>>>(d_data1);
  simpleKernel2<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK,
                  THREADS_PER_BLOCK, 0, stream1>>>(d_data1);

  // Launch kernels on GPU2 using stream2
  checkCudaError(hipSetDevice(1), "Setting device 1");
  simpleKernel1<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK,
                  THREADS_PER_BLOCK, 0, stream2>>>(d_data2);
  simpleKernel2<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK,
                  THREADS_PER_BLOCK, 0, stream2>>>(d_data2);

  // Synchronize the streams
  checkCudaError(hipStreamSynchronize(stream1), "Stream1 synchronize");
  checkCudaError(hipStreamSynchronize(stream2), "Stream2 synchronize");

  // Clean up
  checkCudaError(hipStreamDestroy(stream1), "Destroying stream1");
  checkCudaError(hipStreamDestroy(stream2), "Destroying stream2");
}

int main() {
  float *h_data;
  float *d_data1, *d_data2;

  // Allocate pinned host memory
  checkCudaError(hipHostMalloc((void **)&h_data, N * sizeof(float), hipHostMallocDefault),
                 "Allocating pinned host memory");

  // Initialize host data
  for (int i = 0; i < N; i++) {
    h_data[i] = static_cast<float>(i);
  }

  // Allocate memory on GPU1
  checkCudaError(hipSetDevice(0), "Setting device 0");
  checkCudaError(hipMalloc((void **)&d_data1, N * sizeof(float)),
                 "Allocating d_data1");

  // Allocate memory on GPU2
  checkCudaError(hipSetDevice(1), "Setting device 1");
  checkCudaError(hipMalloc((void **)&d_data2, N * sizeof(float)),
                 "Allocating d_data2");

  // Measure time for single stream version
  auto start = std::chrono::high_resolution_clock::now();
  singleStreamTransferAndCompute(h_data, d_data1, d_data2);
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = end - start;
  std::cout << "Single stream time: " << elapsed.count() << " seconds"
            << std::endl;

  // Clean up device memory
  checkCudaError(hipFree(d_data1), "Freeing d_data1");
  checkCudaError(hipFree(d_data2), "Freeing d_data2");

  // Allocate memory again for multi-stream version
  checkCudaError(hipSetDevice(0), "Setting device 0");
  checkCudaError(hipMalloc((void **)&d_data1, N * sizeof(float)),
                 "Allocating d_data1 again");
  checkCudaError(hipSetDevice(1), "Setting device 1");
  checkCudaError(hipMalloc((void **)&d_data2, N * sizeof(float)),
                 "Allocating d_data2 again");

  // Measure time for multi-stream version
  start = std::chrono::high_resolution_clock::now();
  multiStreamTransferAndCompute(h_data, d_data1, d_data2);
  end = std::chrono::high_resolution_clock::now();
  elapsed = end - start;
  std::cout << "Multi-stream time: " << elapsed.count() << " seconds"
            << std::endl;

  // Clean up
  checkCudaError(hipFree(d_data1), "Freeing d_data1");
  checkCudaError(hipFree(d_data2), "Freeing d_data2");
  checkCudaError(hipHostFree(h_data), "Freeing pinned host memory");

  return 0;
}