
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdio.h>

__device__ long factorial_recursive(long n) {
  if (n <= 1)
    return 1;
  else
    return n + factorial_recursive(n - 1);
}

__device__ long factorial_iterative(long n) {
  long result = 0;
  for (long i = 1; i <= n; ++i) {
    result += i;
  }
  return result;
}

__global__ void compute_factorial_recursive(long n, long *result) {
  result[0] = factorial_recursive(n);
}

__global__ void compute_factorial_iterative(long n, long *result) {
  result[0] = factorial_iterative(n);
}

int main() {
  long n;
  long *result;
  hipMallocManaged(&result, sizeof(long));

  hipEvent_t start, stop;
  float timeRecursive, timeIterative;

  for (n = 10; n <= 1000000; n *= 10) {
    printf("current size is %d\n", n);
    // Initialize CUDA events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Measure recursive version
    hipEventRecord(start);
    compute_factorial_recursive<<<1, 1>>>(n, result);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&timeRecursive, start, stop);
    printf("Recursive Factorial of %d is %d, Time: %f ms\n", n, result[0],
           timeRecursive);

    // Measure iterative version
    hipEventRecord(start);
    compute_factorial_iterative<<<1, 1>>>(n, result);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&timeIterative, start, stop);
    printf("Iterative Factorial of %d is %d, Time: %f ms\n", n, result[0],
           timeIterative);

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }
  hipFree(result);

  return 0;
}
