#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel using Cache-All (CA) load
__global__ void load_ca(int *data, int n) {
  int cur = 0;
  int sum = 0;
  for (int i = 0; i < n; i++) {
    // asm("ld.ca.s32 %0, [%1];" : "=r"(cur) : "l"(&data[i]));
    cur = __ldg(&data[i]);
    sum += cur;
    __syncwarp();
    // __threadfence_block();
  }
  data[0] = sum;
}

// Kernel using Cache-Global (CG) load
__global__ void load_cg(volatile int *data, int n) {
  int cur = 0;
  int sum = 0;
  for (int i = 0; i < n; i++) {
    asm("ld.cg.s32 %0, [%1];" : "=r"(cur) : "l"(&data[i]));
    sum += cur;
    __syncwarp();
    // __threadfence_block();
  }
  data[0] = sum;
}

__global__ void load_cv(volatile int *data, int n) {
  int cur = 0;
  int sum = 0;
  for (int i = 0; i < n; i++) {
    asm("ld.cv.s32 %0, [%1];" : "=r"(cur) : "l"(&data[i]));
    sum += cur;
    __syncwarp();
    // __threadfence_block();
  }
  data[0] = sum;
}

// Kernel using atomicCAS to perform operations on different memory locations
__global__ void atomic_cas_kernel(int *data, int n) {
  int sum = 0;
  for (int i = 0; i < n; i++) {
    sum += atomicCAS(&data[i], 0, 1); // Swap 0 to 1 at each index
    // __syncwarp();
    // __threadfence_block();
  }
  data[0] = sum;
}

// Kernel using atomicMin to perform operations on different memory locations
__global__ void atomic_min_kernel(int *data, int n) {
  int sum = 0;
  for (int i = 0; i < n; i++) {
    sum += atomicMin(&data[i], 1); // Set the minimum to 1 at each index
    __syncwarp();
    // __threadfence_block();
  }

  data[0] = sum;
}

// Kernel for sequential read and write increment
__global__ void read_write_increment_kernel(int *data, int n) {
  int val = 0;
  for (int i = 0; i < n; i++) {
    // data[i] = 1;
    // asm("st.cg.s32 [%0], %1;"
    //     : // No outputs
    //     : "l"(data + 0),
    //       "r"(val) // Inputs - "l" for a 64-bit address, "r"
    //                // for a 32-bit integer
    //     : "memory" // Tells the compiler that memory is being
    //                // modified
    // );
    asm("st.wt.s32 [%0], %1;"
        : // No outputs
        : "l"(data + 0),
          "r"(val) // Inputs - "l" for a 64-bit address, "r"
                   // for a 32-bit integer
        : "memory" // Tells the compiler that memory is being
                   // modified
    );
    __syncwarp();
    // __threadfence();
    val++;
  }
}

void measureKernelPerformance(int *d_data, int n, void (*kernel)(int *, int),
                              const char *kernelName) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;

  hipEventRecord(start);
  kernel<<<1, 1>>>(d_data, n); // Launch with one block of one thread
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("%s kernel execution time: %.5f ms\n", kernelName, milliseconds);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}

void measureKernelPerformanceV(int *d_data, int n,
                               void (*kernel)(volatile int *, int),
                               const char *kernelName) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;

  hipEventRecord(start);
  kernel<<<1, 1>>>(d_data, n); // Launch with one block of one thread
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("%s kernel execution time: %.5f ms\n", kernelName, milliseconds);

  hipEventDestroy(start);
  hipEventDestroy(stop);
}

int main() {
  const int n = 1000 * 1000 * 10; // Number of elements
  int *d_data, *d_result;

  // Allocate memory
  hipMalloc(&d_data, n * sizeof(int));

  // Initialize data
  int *h_data = new int[n];
  for (int i = 0; i < n; ++i) {
    h_data[i] = i;
  }
  hipMemcpy(d_data, h_data, n * sizeof(int), hipMemcpyHostToDevice);

  // Measure each kernel's performance
  measureKernelPerformance(d_data, n, load_ca, "Cache-All (CA)");
  measureKernelPerformanceV(d_data, n, load_cg, "Cache-Global (CG)");
  measureKernelPerformanceV(d_data, n, load_cv, "Global (CV)");
  measureKernelPerformance(d_data, n, atomic_cas_kernel, "Atomic CAS");
  measureKernelPerformance(d_data, n, atomic_min_kernel, "Atomic Min");
  measureKernelPerformance(d_data, n, read_write_increment_kernel,
                           "Read-Write Increment");

  // Cleanup
  hipFree(d_data);
  hipFree(d_result);
  delete[] h_data;

  return 0;
}
