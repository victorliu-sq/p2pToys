#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel using atomicCAS to perform operations on different memory locations
__global__ void atomic_cas_kernel(int *data, int n) {
  if (threadIdx.x == 0) {
    for (int i = 0; i < n; i++) {
      atomicCAS(&data[i], 0, 1); // Swap 0 to 1 at each index
    }
  }
}

// Kernel using atomicMin to perform operations on different memory locations
__global__ void atomic_min_kernel(int *data, int n) {
  if (threadIdx.x == 0) {
    for (int i = 0; i < n; i++) {
      atomicMin(&data[i], 1); // Set the minimum to 1 at each index
    }
  }
}

int main() {
  const int n = 1024 * 1024; // Number of operations
  int *d_data;
  hipMalloc(&d_data, n * sizeof(int));
  hipMemset(d_data, 0, n * sizeof(int));

  // Setup CUDA events for timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds;

  // Launch atomicCAS kernel with only one thread and measure time
  hipEventRecord(start);
  atomic_cas_kernel<<<1, 1>>>(d_data, n); // One block, one thread
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Time for atomicCAS on different memory locations with one thread: "
         "%.5f ms\n",
         milliseconds);

  // Reset data for the next test
  hipMemset(d_data, 0, n * sizeof(int));

  // Launch atomicMin kernel with only one thread and measure time
  hipEventRecord(start);
  atomic_min_kernel<<<1, 1>>>(d_data, n); // One block, one thread
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Time for atomicMin on different memory locations with one thread: "
         "%.5f ms\n",
         milliseconds);

  // Cleanup
  hipFree(d_data);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
