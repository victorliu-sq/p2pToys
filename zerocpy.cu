#include <hip/hip_runtime.h>
#include <iostream>

__global__ void copyKernel(int *d_src, int *d_dst, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n) {
    d_dst[idx] = d_src[idx];
  }
}

__global__ void zeroCopyKernel(int *d_src, int *d_dst, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n) {
    d_dst[idx] = d_src[idx];
  }
}

void scenario1(int *h_list, int n) {
  int *d_list1, *d_list2;
  size_t size = n * sizeof(int);

  hipMalloc((void **)&d_list1, size);
  hipMalloc((void **)&d_list2, size);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  // Copy list from host to device
  hipMemcpy(d_list1, h_list, size, hipMemcpyHostToDevice);

  // Launch kernel to copy list to another device memory
  int threadsPerBlock = 256;
  int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
  copyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_list1, d_list2, n);

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  std::cout << "Scenario 1 time: " << milliseconds << " ms" << std::endl;

  hipFree(d_list1);
  hipFree(d_list2);
  hipEventDestroy(start);
  hipEventDestroy(stop);
}

void scenario2(int *h_list, int n) {
  int *d_list2;
  int *h_pinned_list;
  size_t size = n * sizeof(int);

  hipHostAlloc((void **)&h_pinned_list, size, hipHostMallocMapped);
  hipHostGetDevicePointer((void **)&d_list2, h_pinned_list, 0);

  for (int i = 0; i < n; ++i) {
    h_pinned_list[i] = h_list[i];
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  // Launch kernel to copy list to another device memory
  int threadsPerBlock = 256;
  int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
  zeroCopyKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_list2, d_list2, n); // Self copy for demonstration

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  std::cout << "Scenario 2 time: " << milliseconds << " ms" << std::endl;

  hipHostFree(h_pinned_list);
  hipEventDestroy(start);
  hipEventDestroy(stop);
}

int main() {
  int n = 1 << 20; // Example size
  int *h_list = new int[n];

  // Initialize the list
  for (int i = 0; i < n; ++i) {
    h_list[i] = i;
  }

  scenario1(h_list, n);
  scenario2(h_list, n);

  delete[] h_list;
  return 0;
}