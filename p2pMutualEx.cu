#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK(func)                                                       \
  do {                                                                         \
    hipError_t rt = (func);                                                   \
    if (rt != hipSuccess) {                                                   \
      std::cout << "API call failure \"" #func "\" with " << rt << " at "      \
                << __FILE__ << ":" << __LINE__ << std::endl;                   \
      throw;                                                                   \
    }                                                                          \
  } while (0);


// GPU1 increments elements by 1
__global__ void incrementByOneKernel(int *data, int size) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < size) {
        data[idx] += 1;
    }
    if (idx == 0) { // Print from a single thread to avoid clutter
        printf("GPU1 incremented data.\n");
    }
}

// GPU2 increments elements by 2
__global__ void incrementByTwoKernel(int *data, int size) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < size) {
        data[idx] += 2;
    }
    if (idx == 0) { // Print from a single thread to avoid clutter
        printf("GPU2 incremented data.\n");
    }
}

int main() {
    int *d_gpu0Data, *d_gpu1Data;
    int size = 1024; // Example size
    size_t bytes = size * sizeof(int);

    // Initialize CUDA P2P
    int canAccessPeer01, canAccessPeer10;
    hipDeviceCanAccessPeer(&canAccessPeer01, 0, 1);
    hipDeviceCanAccessPeer(&canAccessPeer10, 1, 0);
    if (!(canAccessPeer01 && canAccessPeer10)) {
        std::cerr << "P2P access not supported between the GPUs.\n";
        return EXIT_FAILURE;
    }

    hipSetDevice(0);
    CUDA_CHECK(hipDeviceEnablePeerAccess(1, 0));
    hipMalloc(&d_gpu0Data, bytes);

    hipSetDevice(1);
    CUDA_CHECK(hipDeviceEnablePeerAccess(0, 0));
    hipMalloc(&d_gpu1Data, bytes);

    // GPU1 increments data on GPU2 by 1
    hipSetDevice(1); // Switch to GPU1's context to access GPU2's memory
    incrementByOneKernel<<<(size + 255) / 256, 256>>>(d_gpu1Data, size);

    CUDA_CHECK(hipGetLastError());

    // GPU2 increments data on GPU1 by 2
    hipSetDevice(0); // Switch to GPU2's context to access GPU1's memory
    incrementByTwoKernel<<<(size + 255) / 256, 256>>>(d_gpu0Data, size);

    CUDA_CHECK(hipGetLastError());

    // Wait for GPUs to finish
    hipSetDevice(0);
    hipDeviceSynchronize();
    hipSetDevice(1);
    hipDeviceSynchronize();

    // Cleanup
    hipSetDevice(0);
    hipFree(d_gpu0Data);
    hipDeviceDisablePeerAccess(1);

    hipSetDevice(1);
    hipFree(d_gpu1Data);
    hipDeviceDisablePeerAccess(0);

    return 0;
}
