#include <hip/hip_runtime.h>
#include <stdio.h>

// __global__ void memory_load_test(volatile int *global_data, int n) {
//   int local; // Adjust size based on maximum expected `n`
//   // int sum = 0;
//   for (int i = 0; i < n; i++) {
//     local = global_data[i]; // Load data from global to local memory
//     // sum += local;
//   }
// }

__global__ void memory_load_test(int *global_data, int *indices, int n,
                                 int *sum) {
  int local;
  *sum = 0;
  for (int i = 0; i < n; i++) {
    local = global_data[indices[i]]; // Load data from global memory at random
                                     // indices
    *sum += local;
  }
}

int main() {
  int *d_data, *d_indices, *d_sum;
  int *h_indices, sum;

  for (int n = 1024; n <= 1024 * 1024; n <<= 1) {
    size_t bytes = n * sizeof(int);

    // Allocate memory on host and device
    hipMalloc(&d_data, bytes);
    hipMalloc(&d_indices, bytes);
    hipMalloc(&d_sum, sizeof(int));
    int *h_data = (int *)malloc(bytes);
    h_indices = (int *)malloc(bytes);

    // Initialize data on host
    for (int i = 0; i < n; i++) {
      h_data[i] = i; // Sequential numbers as data
    }

    // Generate random indices on host
    for (int i = 0; i < n; i++) {
      h_indices[i] = rand() % n;
    }

    // Copy data from host to device
    hipMemcpy(d_data, h_data, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_indices, h_indices, bytes, hipMemcpyHostToDevice);

    // Setup timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Launch kernel and measure time
    hipEventRecord(start);
    memory_load_test<<<1, 1>>>(d_data, d_indices, n,
                               d_sum); // Using 1 block of 1 thread
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipDeviceSynchronize();

    // Copy back the sum to check correctness if necessary
    hipMemcpy(&sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken to load %d integers randomly from global to local "
           "memory: %.3f ms\n",
           n, milliseconds);

    // Cleanup
    hipFree(d_data);
    hipFree(d_indices);
    hipFree(d_sum);
    free(h_data);
    free(h_indices);
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }

  return 0;
}
