#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void DoubleLoad(volatile int *global_data, int *indices, int n,
                           int *d_sum) {
  int local;
  int sum = 0;
  int idx;
  for (int i = 0; i < n; i++) {
    idx = indices[i];
    local = global_data[idx]; // Random access to global_data based on
                              // shuffled indices
    sum += local;
  }
  *d_sum = sum;
}

__global__ void SingleLoad(volatile int *global_data, int *indices, int n,
                           int *d_sum) {
  int local;
  int sum = 0;
  int idx;

  for (int i = 0; i < n; i++) {
    idx = indices[i];
    local = global_data[idx]; // Random access to global_data based on
                              // shuffled indices
    // local = global_data[i]; // Linear access to global_data using indices
    sum += local;
  }
  *d_sum = sum;
}

int main() {
  int *d_data, *d_indices, *d_sum;
  int *h_indices, sum;

  for (int n = 1024; n <= 16 * 16 * 1024 * 1024; n <<= 1) {
    size_t bytes = n * sizeof(int);

    // Allocate memory on host and device
    hipMalloc(&d_data, bytes);
    hipMalloc(&d_indices, bytes);
    hipMalloc(&d_sum, sizeof(int));
    int *h_data = (int *)malloc(bytes);
    h_indices = (int *)malloc(bytes);

    // Initialize data on host
    for (int i = 0; i < n; i++) {
      h_data[i] = i;             // Sequential numbers as data
      h_indices[i] = rand() % n; // Random indices for accessing data
    }

    // Copy data from host to device
    hipMemcpy(d_data, h_data, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_indices, h_indices, bytes, hipMemcpyHostToDevice);

    // Setup timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Launch kernel for random access and measure time
    hipEventRecord(start);
    DoubleLoad<<<1, 1>>>(d_data, d_indices, n, d_sum);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    // cudaMemcpy(&sum, d_sum, sizeof(int), cudaMemcpyDeviceToHost);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Double Access: Time taken to load %d integers: %.3f ms\n", n,
           milliseconds);

    // Linear access
    for (int i = 0; i < n; i++) {
      h_indices[i] = i; // Random indices for accessing data
    }
    hipMemcpy(d_indices, h_indices, bytes, hipMemcpyHostToDevice);
    // Launch kernel for linear access and measure time
    hipEventRecord(start);
    SingleLoad<<<1, 1>>>(d_data, d_indices, n, d_sum);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    // cudaMemcpy(&sum, d_sum, sizeof(int), cudaMemcpyDeviceToHost);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Single Access: Time taken to load %d integers: %.3f ms\n", n,
           milliseconds);
    printf("-------------------------------\n");

    // Cleanup
    hipFree(d_data);
    hipFree(d_indices);
    hipFree(d_sum);
    free(h_data);
    free(h_indices);
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }

  return 0;
}
